#include "hip/hip_runtime.h"
__global__ void vector_add_kernel(const torch::Tensor &a, const torch::Tensor &b, torch::Tensor &c){
    dim1 = blockIndex.x * blockDim.x + threadIdx.x;
    dim2 = blockIndex.y * blockDim.y + threadIdx.y;
    c[dim1][dim2] = a[dim1][dim2] + b[dim1][dim2];
}

torch::Tensor vector_add(const torch::Tensor &a, const torch::Tensor &b, torch::Tensor &c){
    return vector_add_kernel(a, b, c);
}

std::vector<torch::Tensor> vector_add_backward(const torch::Tensor &result){
    return {result, result};
}

