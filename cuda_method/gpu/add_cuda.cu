#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// 通过传递指针进行计算

__global__ void vector_add_kernel(float* a, float* b, float* c, int dim_y, int dim_x){
    auto thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    auto thread_y = blockIdx.y * blockDim.y + threadIdx.y;

    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int i=thread_x;i<dim_x;i = i+stride_x){
        for(int j=thread_y;j<dim_y;j = j + stride_y){
            c[j*dim_x + i] = a[j*dim_x+i] + b[j*dim_x+i];
        }
    }
    
}

// 通过host可以通过传递引用对tensor直接进行修改，但是通过cuda，不能通过直接传递tensor引用的方式，此处有bug
void vector_add(torch::Tensor a, torch::Tensor b, torch::Tensor &c){
    dim3 block_per_grid(1);
    dim3 threads_per_block(10, 10);
    vector_add_kernel<<<block_per_grid, threads_per_block>>>(
        (float*)a.data_ptr<float>(),
        (float*)b.data_ptr<float>(),
        (float*)c.data_ptr<float>(),
        c.size(0),
        c.size(1));
    // for(int i = 0; i< a.size(0);i++){
    //     // 通过指针访问
    //     std::cout << *((float*)a.data_ptr<float>()+i) <<std::endl;
    //     std::cout << *((float*)b.data_ptr<float>()+i) <<std::endl;
    // }

    return;
}

