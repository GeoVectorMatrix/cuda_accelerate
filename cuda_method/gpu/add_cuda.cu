#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>


__global__ void vector_add_kernel(const float* a, const float* b, float* c){
    auto dim1 = blockIdx.x * blockDim.x + threadIdx.x;
    auto dim2 = blockIdx.y * blockDim.y + threadIdx.y;
    printf("hello! %d, %d\n", dim1, dim2);
    // printf(c);
    printf("\n");
    // c[dim1][dim2] = a[dim1][dim2] + b[dim1][dim2];
}

// 通过host可以通过传递引用对tensor直接进行修改，但是通过cuda，不能通过直接传递tensor引用的方式，此处有bug
void vector_add(torch::Tensor a, torch::Tensor b, torch::Tensor &c){
    dim3 block_per_grid(2, 2);
    dim3 threads_per_block(2, 2);
    vector_add_kernel<<<block_per_grid, threads_per_block>>>(
        (float*)a.data_ptr(),
        (float*)b.data_ptr(),
        (float*)c.data_ptr());
    std::cout << c.size(0) << std::endl;
    for(int i = 0; i< a.size(0);i++){
        std::cout << (float*)a.data_ptr()+i <<std::endl;
        std::cout << (float*)b.data_ptr()+i <<std::endl;
    }
    // std::cout << c << std::endl;
    return;
}

