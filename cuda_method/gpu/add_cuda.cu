#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
// __global__ void vector_add_kernel(() a, torch::Tensor b, torch::Tensor c){
//     auto dim1 = blockIdx.x * blockDim.x + threadIdx.x;
//     auto dim2 = blockIdx.y * blockDim.y + threadIdx.y;
//     c[dim1][dim2] = a[dim1][dim2] + b[dim1][dim2];
// }

void vector_add(torch::Tensor a, torch::Tensor b, torch::Tensor c){
    // dim3 block_per_grid(2, 2);
    // dim3 threads_per_block(2, 2);
    auto a_data = a.data();
    auto b_data = b.data();
    auto c_data = c.data();
    std::cout << a_data << b_data << c_data << std::endl;
    // vector_add_kernel<<<block_per_grid, threads_per_block>>>(a_data, b_data, c_data);
    return;
}

