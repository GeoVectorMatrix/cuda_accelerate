#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// 通过传递指针进行计算

__global__ void vector_add_kernel(float* a, float* b, float* c, int dim_x, int dim_y){
    auto dim1 = blockIdx.x * blockDim.x + threadIdx.x;
    auto dim2 = blockIdx.y * blockDim.y + threadIdx.y;
    if((dim2*dim_y + dim1)<dim_x*dim_y){
        c[dim2*dim_y + dim1] = a[dim2*dim_y + dim1] + b[dim2*dim_y + dim1];
    }
}

// 通过host可以通过传递引用对tensor直接进行修改，但是通过cuda，不能通过直接传递tensor引用的方式，此处有bug
void vector_add(torch::Tensor a, torch::Tensor b, torch::Tensor &c){
    dim3 block_per_grid(1);
    dim3 threads_per_block(10, 10);
    vector_add_kernel<<<block_per_grid, threads_per_block>>>(
        (float*)a.data_ptr<float>(),
        (float*)b.data_ptr<float>(),
        (float*)c.data_ptr<float>(),
        c.size(0),
        c.size(1));
    // for(int i = 0; i< a.size(0);i++){
    //     // 通过指针访问
    //     std::cout << *((float*)a.data_ptr<float>()+i) <<std::endl;
    //     std::cout << *((float*)b.data_ptr<float>()+i) <<std::endl;
    // }

    return;
}

