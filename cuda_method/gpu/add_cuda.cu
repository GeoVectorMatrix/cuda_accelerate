#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

__global__ void vector_add_kernel(torch::Tensor a, torch::Tensor b, torch::Tensor c){
    auto dim1 = blockIdx.x * blockDim.x + threadIdx.x;
    auto dim2 = blockIdx.y * blockDim.y + threadIdx.y;
    c[dim1][dim2] = a[dim1][dim2] + b[dim1][dim2];
}

// 通过host可以通过传递引用对tensor直接进行修改，但是通过cuda，不能通过直接传递tensor引用的方式，此处有bug
void vector_add(torch::Tensor a, torch::Tensor b, torch::Tensor &c){
    dim3 block_per_grid(2, 2);
    dim3 threads_per_block(2, 2);
    auto a_data = a.data();
    auto b_data = b.data();
    auto c_data = c.data();
    c[1] = 3;
    std::cout << a_data << b_data << c_data << std::endl;

    float *c_data_ptr = (float*)c_data.data_ptr();
    std::cout << c_data_ptr << std::endl;
    // for(int i = 0 ; i < 10 ;i++){
    //     std::cout << *(c_data_ptr+i) << std::endl;
    // }
    vector_add_kernel<<<block_per_grid, threads_per_block>>>(a_data, b_data, c_data);
    return;
}

