#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// 设置c矩阵的元素
__device__ void setElement(float* matrix, int i, int j, int dim_x, int dim_y, int value){
    matrix[i*dim_y+j] = value;
}

// 获取矩阵的索引为[i][j]的元素
__device__ int getElement(float* matrix, int i, int j, int dim_x, int dim_y){
    return matrix[i*dim_y + j];
}

__global__ void matmul_kernel(float* a, float* b, float* c, int dim_x, int dim_y, int dim_a_x, int dim_a_y, int dim_b_x, int dim_b_y){
    printf("%d, %d, %d, %d, %d, %d", dim_x, dim_y, dim_a_x, dim_a_y, dim_b_x, dim_b_y);
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int i=thread_y; i< dim_y; i = i + stride_y){
        for(int j=thread_x; j < dim_x;j = j + stride_x){
            printf("%d, %d", i, j);
            int value = 0;
            for(int k=0; k< dim_a_y;k++){
                value = value + getElement(a, i, k, dim_a_x, dim_a_y)*getElement(a, k, j, dim_b_x, dim_b_y);
            }
            setElement(c, i, j, dim_x, dim_y, value);
        }
    }
}

void launch_mat_mul(torch::Tensor a, torch::Tensor b, torch::Tensor &c){
    dim3 block_per_grid = (1);
    dim3 threads_per_block = (10, 10);
    float* a_data_ptr = (float*)a.data_ptr<float>();
    float* b_data_ptr = (float*)b.data_ptr<float>();
    float* c_data_ptr = (float*)c.data_ptr<float>();
    int dim_x = c.size(0);
    int dim_y = c.size(1);
    int dim_a_x = a.size(0);
    int dim_a_y = a.size(1);
    int dim_b_x = b.size(0);
    int dim_b_y = b.size(1);
    matmul_kernel<<<block_per_grid, threads_per_block>>>(a_data_ptr, b_data_ptr, c_data_ptr, dim_x, dim_y, dim_a_x, dim_a_y, dim_b_x, dim_b_y);
    return;
}


